#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#define MIN_VALUE (-1e38)
typedef at::Half fp16;
__half *cast(fp16 *ptr){return reinterpret_cast<__half *>(ptr);}

#define MM8_ONE_JSPLIT 24
#define MM8_ONE_TILE 1024




__global__ void kernel_mm_seq_fp32i8(
    const int B, const int N, const int M,
    const float *__restrict__ const x, const int x_stride,
    const uint8_t *__restrict__ const w, const int w_stride,
    const float *__restrict__ const mx,
    const float *__restrict__ const rx,
    const float *__restrict__ const my,
    const float *__restrict__ const ry,
    float *__restrict__ const y, const int y_stride) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < B && k < M) {
        float y_local = 0;
        for (int j = 0; j < N; ++j)
        {
            y_local += x[i * x_stride + j] * ((float(w[j * w_stride + k]) + 0.5f) * rx[k] * ry[j] + mx[k] + my[j]);
        }
        y[i * y_stride + k] = y_local;
    }
}


__global__ void kernel_mm_one_fp32i8(
    const int N, const int M,
    const float *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    const float *__restrict__ const mx,
    const float *__restrict__ const rx,
    const float *__restrict__ const my,
    const float *__restrict__ const ry,
    float *__restrict__ const y)
{

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M)
    {
        float y_local = 0;
        for (int j = j0; j < j1; ++j)
        {
            y_local += x[j] * ((float(w[j * w_stride + k]) + 0.5f) * rx[k] * ry[j] + mx[k] + my[j]);
        }
        atomicAdd(&y[k], y_local);
    }
}


__global__ void kernel_mm_seq_fp16i8(
    const int B, const int N, const int M,
    const __half *__restrict__ const x, const int x_stride,
    const uint8_t *__restrict__ const w, const int w_stride,
    const __half *__restrict__ const mx,
    const __half *__restrict__ const rx,
    const __half *__restrict__ const my,
    const __half *__restrict__ const ry,
    __half *__restrict__ const y, const int y_stride)
{

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < B && k < M)
    {
        float y_local = 0;
        for (int j = 0; j < N; ++j)
        {
            y_local += __half2float(x[i * x_stride + j]) * ((float(w[j * w_stride + k]) + 0.5f) * __half2float(rx[k]) * __half2float(ry[j]) + __half2float(mx[k]) + __half2float(my[j]));
        }
        y[i * y_stride + k] = __float2half(y_local);
    }
}


__global__ void kernel_mm_one_fp16i8(
    const int N, const int M,
    const __half *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    const __half *__restrict__ const mx,
    const __half *__restrict__ const rx,
    const __half *__restrict__ const my,
    const __half *__restrict__ const ry,
    float *__restrict__ const y)
{

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M)
    {
        float y_local = 0;
        for (int j = j0; j < j1; ++j)
        {
            y_local += __half2float(x[j]) * ((float(w[j * w_stride + k]) + 0.5f) * __half2float(rx[k]) * __half2float(ry[j]) + __half2float(mx[k]) + __half2float(my[j]));
        }
        atomicAdd(&y[k], y_local);
    }
}






template <typename F>
void cuda_mm8_seq(int B, int N, int M, F *x, int x_stride, uint8_t *w, int w_stride, F *mx, F *rx, F *my, F *ry, F *y, int y_stride);

template <>
void cuda_mm8_seq<float>(int B, int N, int M, float *x, int x_stride, uint8_t *w, int w_stride, float *mx, float *rx, float *my, float *ry, float *y, int y_stride)
{
    dim3 blockSize(1, 128);
    dim3 gridSize((B + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm_seq_fp32i8<<<gridSize, blockSize>>>(B, N, M, x, x_stride, w, w_stride, mx, rx, my, ry, y, y_stride);
}


template <>
void cuda_mm8_seq<fp16>(int B, int N, int M, fp16 *x, int x_stride, uint8_t *w, int w_stride, fp16 *mx, fp16 *rx, fp16 *my, fp16 *ry, fp16 *y, int y_stride)
{
    dim3 blockSize(1, 128);
    dim3 gridSize((B + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm_seq_fp16i8<<<gridSize, blockSize>>>(B, N, M, cast(x), x_stride, w, w_stride, cast(mx), cast(rx), cast(my), cast(ry), cast(y), y_stride);
}


template <typename F>
void cuda_mm8_one(int N, int M, F *x, uint8_t *w, int w_stride, F *mx, F *rx, F *my, F *ry, float *y);

template <>
void cuda_mm8_one<float>(int N, int M, float *x, uint8_t *w, int w_stride, float *mx, float *rx, float *my, float *ry, float *y)
{
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm_one_fp32i8<<<gridSize, blockSize>>>(N, M, x, w, w_stride, mx, rx, my, ry, y);
}


template <>
void cuda_mm8_one<fp16>(int N, int M, fp16 *x, uint8_t *w, int w_stride, fp16 *mx, fp16 *rx, fp16 *my, fp16 *ry, float *y)
{
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm_one_fp16i8<<<gridSize, blockSize>>>(N, M, cast(x), w, w_stride, cast(mx), cast(rx), cast(my), cast(ry), y);
}
